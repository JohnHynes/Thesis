#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>

#include "constants.hpp"
#include "types.hpp"

#include "TOMLLoader.hpp"
#include "Util.hpp"

using node_ptr = hittable *;

__host__ __device__ inline bool find_closest_hit(const scene *world, node_ptr* stack, ray &r, num t_min,
                                          num t_max, hit_record &hitrec) {
  // Allocate thread-local stack
  node_ptr *stack_ptr = stack;

  // Initialize stack
  *stack_ptr++ = NULL;

  // Initialize local variables
  hit_record temp_hitrec;
  num closest_seen = t_max;
  bool has_hit = false;

  // Traverse tree starting from the root
  node_ptr node = world->hittables;
  do {
    if (node->hit(r, t_min, closest_seen, temp_hitrec)) {
      // node was hit, test for leaf
      if (node->id != hittable_id::BoundingArrayNode) {
        // node is a leaf
        if (temp_hitrec.t < closest_seen) {
          closest_seen = temp_hitrec.t;
          hitrec = temp_hitrec;
        }
        has_hit = true;
      } else {
        // node is not a leaf, push left and right children onto stack.
        *stack_ptr++ = world->hittables + node->as_bounding_array_node().left;
        *stack_ptr++ = world->hittables + node->as_bounding_array_node().right;
      }
    }
    // pop node off stack
    node = *--stack_ptr;

  } while (node != NULL);

  return has_hit;
}

__host__ __device__ color trace_ray(RandomState *state, node_ptr* stack, ray r, color background_color,
                const scene *world, int depth) {
  hit_record rec;
  color attenuation;
  color result_color(1, 1, 1);

  while (depth > 0) {
    // Test bvh for a hit
    if (find_closest_hit(world, stack, r, 0.0001f, infinity, rec)) {
      if (world->materials[rec.mat_idx].scatter((RandomState *)state, r, rec,
                                                attenuation, r)) {
        result_color *= attenuation;
      } else {
        result_color *= world->materials[rec.mat_idx].emit();
        break;
      }
    } else {
      result_color *= background_color;
      break;
    }
    --depth;
  }
  return result_color;
}

__global__ void
__launch_bounds__(256, 4) // gimme 1024 threads
make_image (int seed, int samples_per_pixel, color background_color,
            scene* world, camera* cam, int max_depth, color* d_image)
{
  extern __shared__ char shm[];


  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  const int width = blockDim.x * gridDim.x;
  const int height = blockDim.y * gridDim.y;

  const int id = i + width * j;


  RandomStateGPU state;
  hiprand_init (seed, id, 0, &state);
  RandomState* rngState = (RandomState*)&state;

  // shared memory

  scene local_world;
  local_world.material_count = world->material_count;
  local_world.object_count = world->object_count;
  local_world.hittables_size = world->hittables_size;
  local_world.materials = (material*)shm;
  local_world.hittables = (hittable*)(local_world.materials + local_world.material_count);

  const int local_id = threadIdx.x + blockDim.x * threadIdx.y;
  if (local_id < local_world.material_count) {
    local_world.materials[local_id] = world->materials[local_id];
  }
  if (local_id < local_world.hittables_size) {
    local_world.hittables[local_id] = world->hittables[local_id];
  }

  __syncthreads(); // all threads must wait until the information has been loaded

  // allocating thread-local stack
  node_ptr stack[32];

  color pixel_color (0.0f, 0.0f, 0.0f);
  for (int s = 0; s < samples_per_pixel; ++s)
  {
    num u = (i + random_positive_unit (rngState)) / (width - 1);
    num v = (j + random_positive_unit (rngState)) / (height - 1);
    ray r = cam->get_ray (rngState, u, v);
    pixel_color += trace_ray (rngState, stack, r, background_color, &local_world, max_depth);
  }
  d_image[j * width + i] = pixel_color;
}

int
main (int argc, char* argv[])
{
  string filename{"scene.toml"};
  string output{"image.ppm"};
  int seed{1337};
  if (argc > 1)
  {
    filename = argv[1];
  }
  if (argc > 2)
  {
    output = argv[2];
  }
  if (argc > 3)
  {
    seed = atoi(argv[3]);
  }

  std::cerr << "Command: " << argv[0] << ' ' << filename << ' ' << output << ' ' << seed << '\n';

  const auto scene_data = toml::parse (filename);
  auto [samples_per_pixel, max_depth, image_width, image_height, background_color] = loadParams (scene_data);
  scene world = loadScene (scene_data);
  camera cam = loadCamera (scene_data);

  scene* d_world = world.copy_to_device ();
  camera* d_cam = cam.copy_to_device ();

  color *image, *d_image;
  int num_pixels = image_width * image_height;
  image = new color[num_pixels];
  CUDA_CALL (hipMalloc ((void**)&d_image, num_pixels * sizeof (color)));

  // Declaring block dimensions
  dim3 threads{16, 16};
  dim3 blocks{image_width / threads.x, image_height / threads.y};
  // request enough shared memory to hold all of the materials and hittables
  int shmSize = sizeof(material) * world.material_count + sizeof(hittable) * world.hittables_size;

  // Rendering Image on device
  make_image<<<blocks, threads, shmSize>>> (seed, samples_per_pixel, background_color, d_world, d_cam, max_depth, d_image);
  CUDA_CALL (hipDeviceSynchronize ());

  //d_world->free_device();
  //CUDA_CALL (hipFree(d_cam));

  // Copying 2D buffer from device to host
  CUDA_CALL (hipMemcpy (image, d_image, num_pixels * sizeof (color), hipMemcpyDeviceToHost));
  //CUDA_CALL (hipFree(d_image));

  std::ofstream ofs{output};
  // Outputting Render Data
  ofs << "P3\n";
  ofs << image_width << " " << image_height << "\n";
  ofs << 255 << "\n";

  for (int j = image_height - 1; j >= 0; --j)
  {
    for (int i = 0; i < image_width; ++i)
    {
      auto pixel_color = image[j * image_width + i];
      write_color (ofs, pixel_color, samples_per_pixel);
    }
  }

  delete[] image;
}
